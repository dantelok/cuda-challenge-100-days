#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000
#define BLOCK_SIZE 256

// c is a scalar; where a and b is 2 vectors
__global__ void naive_dot_product(double *a, double *b, double *c, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < n) {
        // (*c) += a[id] * b[id];  // race condition, might get wrong results
        atomicAdd(c, a[id] * b[id]);  // Fix race condition
    }
}

void init_vector(double *vector, int n) {
    for (int i=0; i < n; i++) {
        vector[i] = (double) rand() / RAND_MAX;
    }
}

int main() {
    double *host_a, *host_b, *host_c, *device_a, *device_b, *device_c;

    // Get size
    size_t size = N * sizeof(double);

    // Allocate memory to the host
    host_a = (double*)malloc(size);
    host_b = (double*)malloc(size);
    host_c = (double*)malloc(sizeof(double));

    // Allocate memory to the device
    hipMalloc(&device_a, size);
    hipMalloc(&device_b, size);
    hipMalloc(&device_c, sizeof(double));

    // init value
    init_vector(host_a, N);
    init_vector(host_b, N);
    // *device_c = 0;  // Incorrect memory access: Cannot assign device memory directly from host 

    // Copy from host to device
    hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);

    // Init c = 0
    double zero = 0.0;
    hipMemcpy(device_c, &zero, sizeof(double), hipMemcpyHostToDevice);
    
    // Get grid num
    int nums_block = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Run the Kernel
    naive_dot_product<<<nums_block, BLOCK_SIZE>>>(device_a, device_b, device_c, N);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(host_c, device_c, sizeof(double), hipMemcpyDeviceToHost);

    // Verify Results
    printf("c: %f \n", *host_c);

    // Free Memory
    free(host_a);
    free(host_b);
    free(host_c);
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
}